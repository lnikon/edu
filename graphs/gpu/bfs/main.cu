
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <memory.h>
#include <string.h>
#include <math.h>


#define VCOUNT 5
#define ECOUNT 12

/*
 0 - -- - 1
  -     -
    -  -
      2
    -  -
  -     -
 3 - -- - 4
*/

bool frontierNotEmpty(bool* pFrontier, const int size);
void print(bool* p, const int size, const int unit);

__global__ void bfs_kernel(int* pV, 
                           int* pE, 
                           bool* pF, 
                           bool* pX, 
                           int* pC, 
                           const int vcount, 
                           const int ecount,
                           bool* done)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("tid = %d\n", tid);
    if (tid > VCOUNT) { *done = false; }
    // printf("done = %d\n", *done);

    if (pF[tid] == true && pX[tid] == false)
    {
        printf("tid = %d\n", tid);
        pF[tid] = false;
        pX[tid] = true;
        __syncthreads();

        const int nidStart = pV[tid];
        
        int nidEnd = -1;
        if (tid >= vcount)
        {
            nidEnd = pV[tid + 1];
        }
        else
        {
            nidEnd = ecount;
        }

        // printf("nidStart = %d\n", nidStart);
        // printf("nidEnd = %d\n", nidEnd);

        for (int nid = nidStart; nid < nidEnd; ++nid)
        {
            if (!pX[nid])
            {
                pC[nid] = pC[tid] + 1;
                pF[nid] = true;
                *done = false;
            }
        }
    }
}

int main()
{
    // Allocate space for vertex, edge, frontier, visit and cost arrays
    int*  pVertices  = new int[VCOUNT];
    int*  pEdges     = new int[ECOUNT];
    bool* pFrontier  = new bool[VCOUNT];
    bool* pVisited   = new bool[VCOUNT];
    int*  pCost      = new int[VCOUNT];

    // Initialize arrays
    for (int i = 0; i < VCOUNT; ++i)
    {
        pVertices[i] = i;
        pFrontier[i] = false;
        pVisited[i] = false;
        pCost[i] = INT_MAX;
    }

    pEdges[0] = 1; pEdges[1] = 2;
    pEdges[2] = 0; pEdges[3] = 2;
    pEdges[4] = 0; pEdges[5] = 1; pEdges[6] = 3; pEdges[7] = 4;
    pEdges[8] = 2; pEdges[9] = 4;
    pEdges[10] = 2; pEdges[11] = 3;

    int*  d_pVertices;
    int*  d_pEdges;
    bool* d_pFrontier;
    bool* d_pVisited;
    int*  d_pCost;

    hipMalloc((void **)&d_pVertices,   sizeof(int) * VCOUNT);
    hipMalloc((void **)&d_pEdges,      sizeof(int) * ECOUNT);
    hipMalloc((void **)&d_pFrontier,   sizeof(bool) * VCOUNT);
    hipMalloc((void **)&d_pVisited,    sizeof(bool) * VCOUNT);
    hipMalloc((void **)&d_pCost,       sizeof(int) * VCOUNT);

    int source = 2;
    pFrontier[source] = true;
    pCost[source] = 0;

    hipMemcpy(d_pVertices, pVertices,  sizeof(int) * VCOUNT, hipMemcpyHostToDevice);
    hipMemcpy(d_pEdges,    pEdges,     sizeof(int) * ECOUNT, hipMemcpyHostToDevice);
    hipMemcpy(d_pFrontier, pFrontier,  sizeof(bool) * VCOUNT, hipMemcpyHostToDevice);
    hipMemcpy(d_pVisited,  pVisited,   sizeof(bool) * VCOUNT, hipMemcpyHostToDevice);
    hipMemcpy(d_pCost,     pCost,      sizeof(int) * VCOUNT, hipMemcpyHostToDevice);

    // Actual algorithm code goes here
    int count = 0;
    bool done = true;
    bool* d_done;
    hipMalloc((void**)&d_done, sizeof(bool));
    do
    {
        ++count;
        done = true;
        hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
        bfs_kernel<<<1, VCOUNT>>>(d_pVertices, 
                d_pEdges, 
                d_pFrontier, 
                d_pVisited, 
                d_pCost, 
                VCOUNT, 
                ECOUNT,
                d_done);            
        hipMemcpy(&done, d_done, sizeof(bool), hipMemcpyDeviceToHost);
    } while(!done);

    std::cout << "count: " << count << std::endl;

    delete pVertices;
    delete pEdges;
    delete pFrontier;
    delete pVisited;
    delete pCost;

    hipFree(d_pVertices);
    hipFree(d_pEdges);
    hipFree(d_pFrontier);
    hipFree(d_pVisited);
    hipFree(d_pCost);

    return 0;
}

bool frontierNotEmpty(bool* pFrontier, const int size)
{
    for (int i = 0; i < size; ++i) if (pFrontier[i]) return true;
    return false;
}

void print(bool* p, const int size, const int unit)
{
    for (int i = 0; i < size; ++i) 
    {
        std::cout << p[i] << ' ';
    }
    std::cout << std::endl;
}
